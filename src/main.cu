#include "hip/hip_runtime.h"
#include <SDL3/SDL.h>
#include <hip/hip_runtime.h>
#include <>
#include <vector>
#include <stdexcept>
#include <iostream>
#include <chrono>
#include <thread>
#include <cstdint>
#include <cmath>
#include <cstdio>
#include "common.h"

#define BLOCK_SIZE 256
#define SMOOTH_BLOCK_SIZE 16
#define MAX_STATES 32

__global__ void compute_forces(
    Particle* particles,
    Rule* rules,
    int num_particles,
    int num_states,
	float dt,
    float* force_x,
    float* force_y,
	float* mass,
	float* potential_gain
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_particles) return;

    Particle& p_i = particles[i];
    float fx = 0.0f, fy = 0.0f, fd = 0.0f;
	
    for (int j = 0; j < num_particles; ++j) {
        if (i == j) continue;
        Particle p_j = particles[j];
		Rule rule = rules[p_i.state * MAX_STATES + p_j.state];
        float dx = p_j.x - p_i.x;
        float dy = p_j.y - p_i.y;
        float dist2 = dx * dx + dy * dy;
        float dist = sqrtf(dist2 + 1e-6f);
		float influence = expf(-dist * dist / (2.0 * rule.range * rule.range)) * (1 + rule.power / dist);
		float force = (((mass[p_i.state] + influence) * (mass[p_j.state] + influence)) / mass[p_i.state] * mass[p_j.state]) * rule.attraction * influence;

		fx += dx * force / dist;
		fy += dy * force / dist;
		fd += force / dist / p_j.energy;
    }
	p_i.potential += fd * dt;
	p_i.energy += fd * dt / p_i.energy;
    force_x[i] = fx / mass[p_i.state];
    force_y[i] = fy / mass[p_i.state];
}

__global__ void get_avg(Particle* particles, int num_particles, float *average_energy)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_particles) return;
	Particle& p = particles[i];
	atomicAdd(average_energy, p.energy);
}



__global__ void integrate(
    Particle* particles,
    float* force_x,
    float* force_y,
    int num_particles,
    float dt,
	float max_velocity,
	float* mass
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_particles) return;

    Particle& p = particles[i];
	p.vx += force_x[i] * dt;
	p.vy += force_y[i] * dt;
	float speed = sqrtf(p.vx * p.vx + p.vy * p.vy);
	float max_speed = max_velocity;
	if (speed > max_speed) {
		float limit_scale = max_speed / speed;
		p.vx *= limit_scale;
		p.vy *= limit_scale;
	}
    p.x += p.vx * dt;
    p.y += p.vy * dt;
}

__global__ void update_states(
    Particle* particles,
    int num_particles,
    int num_states,
	float dt,
	float* mass,
	float* average_energy,
	float* target_energy
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_particles) return;

    Particle& p = particles[i];
	if (p.potential > (p.energy * p.energy)) {
		int old_state = p.state;
		p.state = (p.state + 1) % num_states;
		p.energy *= mass[p.state] / mass[old_state];
		p.potential /= mass[p.state] / mass[old_state];
	}
	float energy_adjustment = mass[p.state] + (*target_energy - *average_energy / num_particles);
	p.energy = (1.0f - dt) * p.energy + dt * energy_adjustment;
	p.energy = fmaxf(p.energy, 1e-4f);
}

__device__ __host__ inline void unpackRGBA(uint32_t packed, float& r, float& g, float& b, float& a) {
    r = float((packed >> 24) & 0xFF);
    g = float((packed >> 16) & 0xFF);
    b = float((packed >> 8)  & 0xFF);
    a = float((packed)       & 0xFF);
}

__device__ __host__ inline uint32_t packRGBA(float r, float g, float b, float a) {
    uint32_t R = min(max(int(a + 0.5f), 0), 255);
    uint32_t G = min(max(int(b + 0.5f), 0), 255);
    uint32_t B = min(max(int(g + 0.5f), 0), 255);
    uint32_t A = min(max(int(r + 0.5f), 0), 255);
    return (R << 24) | (G << 16) | (B << 8) | A;
}
static Particle* d_particles = nullptr;
static Particle* d_visible = nullptr;
static Rule* d_rules = nullptr;
static float* d_fx = nullptr;
static float* d_fy = nullptr;
static float* d_average_energy = nullptr;
static float* d_target_energy = nullptr;
static float* d_mass = nullptr;
static float* d_potential_gain = nullptr;
static uint32_t* d_input = nullptr;
static int* d_num_states = nullptr;
void init_cuda_managed_buffers(int num_particles, int num_states, int width, int height) {
    if (!d_particles)
        hipMallocManaged(&d_particles, sizeof(Particle) * num_particles);
	if (!d_visible)
        hipMallocManaged(&d_visible, sizeof(Particle) * num_particles);
    if (!d_rules)
        hipMallocManaged(&d_rules, sizeof(Rule) * MAX_STATES * MAX_STATES);
    if (!d_fx)
        hipMallocManaged(&d_fx, sizeof(float) * num_particles);
    if (!d_fy)
        hipMallocManaged(&d_fy, sizeof(float) * num_particles);
    if (!d_average_energy)
        hipMallocManaged(&d_average_energy, sizeof(float));
    if (!d_target_energy)
        hipMallocManaged(&d_target_energy, sizeof(float));
	if (!d_mass)
		hipMallocManaged(&d_mass, sizeof(float) * MAX_STATES);
	if (!d_input) {
		hipMallocManaged(&d_input, width * height * sizeof(uint32_t));
	}
	if (!d_potential_gain) {
		hipMallocManaged(&d_potential_gain, sizeof(float));
	}
	if (!d_num_states) {
		hipMallocManaged(&d_num_states, sizeof(int));
	}
}

void free_cuda_managed_buffers() {
    if (d_particles) hipFree(d_particles);
	if (d_visible) hipFree(d_visible);
    if (d_rules) hipFree(d_rules);
    if (d_fx) hipFree(d_fx);
    if (d_fy) hipFree(d_fy);
    if (d_average_energy) hipFree(d_average_energy);
    if (d_target_energy) hipFree(d_target_energy);
	if (d_mass) hipFree(d_mass);
	if (d_input) hipFree(d_input);
	if (d_potential_gain) hipFree(d_potential_gain);
	if (d_num_states) hipFree(d_num_states);
	
    d_particles = nullptr;
	d_visible = nullptr;
    d_rules = nullptr;
    d_fx = nullptr;
    d_fy = nullptr;
    d_average_energy = nullptr;
    d_target_energy = nullptr;
	d_mass = nullptr;
	d_input = nullptr;
	d_potential_gain = nullptr;
	d_num_states = nullptr;
}

void step_simulation(int num_particles, int num_states, float dt, float max_velocity, float target_energy, int frameskip, float potential_gain) {
    int blockSize = BLOCK_SIZE;
    int gridSize = (num_particles + blockSize - 1) / blockSize;
	*d_target_energy = target_energy;
	*d_potential_gain = potential_gain;
    // Initialize average energy and target energy in managed memory
	for (int i = 0; i < frameskip; ++i) {
		*d_average_energy = 0.0f;
		compute_forces<<<gridSize, blockSize>>>(d_particles, d_rules, num_particles, num_states, dt, d_fx, d_fy, d_mass, d_potential_gain);
		hipDeviceSynchronize();

		get_avg<<<gridSize, blockSize>>>(d_particles, num_particles, d_average_energy);
		hipDeviceSynchronize();

		integrate<<<gridSize, blockSize>>>(d_particles, d_fx, d_fy, num_particles, dt, max_velocity, d_mass);
		hipDeviceSynchronize();

		update_states<<<gridSize, blockSize>>>(d_particles, num_particles, num_states, dt, d_mass, d_average_energy, d_target_energy);
		hipDeviceSynchronize();
	}
}

__global__ void renderDensityBlend(uint32_t* framebuffer, Particle* particles, int num_particles,
                                   int width, int height, int num_states, float offsetX, float offsetY, float zoom, float particle_size, float* mass, float* target_energy) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    float3 color = make_float3(0, 0, 0);
    float2 pixel = make_float2(x + 0.5f, y + 0.5f);
    float radius = particle_size;

    for (int i = 0; i < num_particles; ++i) {
		Particle p = particles[i];
		int px = (int)((p.x - offsetX - width / 2.0f) * zoom + width / 2.0f);
		int py = (int)((p.y - offsetY - height / 2.0f) * zoom + height / 2.0f);
		if (px >= 0 && px < width && py >= 0 && py < height) {
			float2 pos = make_float2(px, py);
			float dx = pos.x - pixel.x;
			float dy = pos.y - pixel.y;
			float intensity = fmaxf(0.0f, fminf(1.0f, radius / (2.0f * (dx * dx + dy * dy) + radius)));

			// Convert particle state to HSV color
			float h = p.state / (float)num_states;
			float s = fminf(1.0f, p.energy / mass[p.state] / *target_energy / 2) * 3 / 4 + 0.25;
			float v = intensity * (fminf(1.0f, fmaxf(0.0f, 1.0f - p.potential / (p.energy * p.energy))) * 3 / 4 + 0.25);

			// HSV → RGB
			float c = v * s;
			float h6 = h * 6.0f;
			float xcol = c * (1.0f - fabsf(fmodf(h6, 2.0f) - 1.0f));
			float3 rgb;
			if      (h6 < 1) rgb = make_float3(c, xcol, 0);
			else if (h6 < 2) rgb = make_float3(xcol, c, 0);
			else if (h6 < 3) rgb = make_float3(0, c, xcol);
			else if (h6 < 4) rgb = make_float3(0, xcol, c);
			else if (h6 < 5) rgb = make_float3(xcol, 0, c);
			else             rgb = make_float3(c, 0, xcol);
			rgb.x += v - c;
			rgb.y += v - c;
			rgb.z += v - c;
			color.x += rgb.x;
			color.y += rgb.y;
			color.z += rgb.z;
		}
    }

    color.x = fminf(1.0f, color.x);
    color.y = fminf(1.0f, color.y);
    color.z = fminf(1.0f, color.z);

    framebuffer[y * width + x] = packRGBA(
        (uint8_t)(color.x * 255.0f),
        (uint8_t)(color.y * 255.0f),
        (uint8_t)(color.z * 255.0f),
        255);
}


int main(int argc, char* argv[]) {
	
    // Initialize SDL
    if (!SDL_Init(SDL_INIT_VIDEO)) {
        std::cerr << "SDL_Init Error: " << SDL_GetError() << "\n";
        return 1;
    }

    SDL_Window* window = SDL_CreateWindow("Vector Display", width, height, 0);
    if (!window) {
        std::cerr << "SDL_CreateWindow Error: " << SDL_GetError() << "\n";
        SDL_Quit();
        return 1;
    }

    SDL_Renderer* renderer = SDL_CreateRenderer(window, NULL);
    if (!renderer) {
        std::cerr << "SDL_CreateRenderer Error: " << SDL_GetError() << "\n";
        SDL_DestroyWindow(window);
        SDL_Quit();
        return 1;
    }

    SDL_Texture* texture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGBA32, SDL_TEXTUREACCESS_STREAMING, width, height);
    if (!texture) {
        std::cerr << "SDL_CreateTexture Error: " << SDL_GetError() << "\n";
        SDL_DestroyRenderer(renderer);
        SDL_DestroyWindow(window);
        SDL_Quit();
        return 1;
    }
	
    // Create host-side particle vector and rules vector
    std::vector<Particle> particles(num_particles);
    std::vector<Rule> rules(MAX_STATES * MAX_STATES);
	std::vector<float> mass(MAX_STATES);

    // Initialize particles with random positions, velocities, states, and energy
	srand(time(0));
    for (Particle& p : particles) {
		p.x = ((float)rand() / RAND_MAX * width - width / 2) / zoom + width / 2 + offsetX;
		p.y = ((float)rand() / RAND_MAX * height - height / 2) / zoom + height / 2 + offsetY;
		p.vx = ((float)rand() / RAND_MAX - 0.5f);
		p.vy = ((float)rand() / RAND_MAX - 0.5f);
		p.state = rand() % num_states;
		p.energy = 1.0;
		p.potential = (float)rand() / RAND_MAX;
	}

    // Initialize rules - deterministic for stable forces
    for (Rule& r : rules) {
        r.attraction = ((float)rand() / RAND_MAX - 0.5f) * 3.0f;
		r.range = 1.0f + ((float)rand() / RAND_MAX) * 29.0f;
		r.power = ((float)rand() / RAND_MAX) * 2.0f;
    }
	for (float& m : mass) {
		m = ((float)rand() / RAND_MAX) * 499.0f + 1.0f;
	}
	
    // Initialize CUDA managed memory buffers
    init_cuda_managed_buffers(num_particles, num_states, width, height);

    // Copy initial particles and rules into managed buffers
    memcpy(d_particles, particles.data(), sizeof(Particle) * num_particles);
    memcpy(d_rules, rules.data(), sizeof(Rule) * MAX_STATES * MAX_STATES);
	memcpy(d_mass, mass.data(), sizeof(float) * MAX_STATES);

    std::vector<uint32_t> framebuffer(width * height, 0);
	std::vector<uint32_t> accum(width * height, 0);
    bool quit = false;
	bool pause = false;
	bool undither = false;
	int particle_size = 1;
    SDL_Event e;

    while (!quit) {
		while (SDL_PollEvent(&e)) {
			switch (e.type) {
				case SDL_EVENT_QUIT:
					quit = true;
					break;
				case SDL_EVENT_KEY_DOWN:
					switch (e.key.key) {
						case SDLK_R:
							for (Rule& r : rules) {
								r.attraction = ((float)rand() / RAND_MAX - 0.5f) * 3.0f;
								r.range = 1.0f + ((float)rand() / RAND_MAX) * 29.0f;
								r.power = ((float)rand() / RAND_MAX) * 2.0f;
							}
							for (float& m : mass) {
								m = ((float)rand() / RAND_MAX) * 499.0f + 1.0f;
							}
							memcpy(d_mass, mass.data(), sizeof(float) * MAX_STATES);
							memcpy(d_rules, rules.data(), sizeof(Rule) * MAX_STATES * MAX_STATES);
							break;
						case SDLK_B:
							for (Particle& p : particles) {
								p.x = ((float)rand() / RAND_MAX * width - width / 2) / zoom + width / 2 + offsetX;
								p.y = ((float)rand() / RAND_MAX * height - height / 2) / zoom + height / 2 + offsetY;
								p.vx = ((float)rand() / RAND_MAX - 0.5f);
								p.vy = ((float)rand() / RAND_MAX - 0.5f);
								p.state = rand() % num_states;
								p.energy = 1.0;
								p.potential = (float)rand() / RAND_MAX;
							}
							memcpy(d_particles, particles.data(), sizeof(Particle) * num_particles);
							break;
						case SDLK_P:
							zoom = 0.5f;
							offsetX = 0.0f;
							offsetY = 0.0f;
							break;
						case SDLK_MINUS:
						case SDLK_KP_MINUS:
							num_states = std::max(2, num_states - 1);
							for (Particle& p : particles) {
								p.state = std::min(p.state, num_states - 1);
							}
							printf("Changed num_states to %d\n", num_states);
							break;
						case SDLK_EQUALS:
						case SDLK_KP_PLUS:
							num_states = std::min(MAX_STATES, num_states + 1);
							printf("Changed num_states to %d\n", num_states);
							break;
						case SDLK_UP:
							dt *= 1.1f;
							printf("Changed dt to %.7f\n", dt);
							break;
						case SDLK_DOWN:
							dt /= 1.1f;
							printf("Changed dt to %.7f\n", dt);
							break;
						case SDLK_RIGHT:
							max_velocity *= 1.1f;
							printf("Changed max_velocity to %.2f\n", max_velocity);
							break;
						case SDLK_LEFT:
							max_velocity /= 1.1f;
							printf("Changed max_velocity to %.2f\n", max_velocity);
							break;
						case SDLK_PERIOD:
							target_energy *= 1.1f;
							printf("Changed target_energy to %.2f\n", target_energy);
							break;
						case SDLK_COMMA:
							target_energy /= 1.1f;
							printf("Changed target_energy to %.2f\n", target_energy);
							break;
						case SDLK_2:
							frameskip++;
							printf("Changed frameskip to %d\n", frameskip);
							break;
						case SDLK_1:
							frameskip--;
							printf("Changed frameskip to %d\n", frameskip);
							break;
						case SDLK_4:
							potential_gain *= 1.1;
							printf("Changed potential_gain to %.5f\n", potential_gain);
							break;
						case SDLK_3:
							potential_gain /= 1.1;
							printf("Changed potential_gain to %.5f\n", potential_gain);
							break;
						case SDLK_SPACE:
							pause = !pause;
							break;
						case SDLK_G:
							particle_size--;
							printf("Changed particle_size to %d\n", particle_size);
							break;
						case SDLK_H:
							particle_size++;
							printf("Changed particle_size to %d\n", particle_size);
							break;
					}
					break;
				case SDL_EVENT_MOUSE_WHEEL:
					if (e.wheel.y > 0) zoom *= 1.1f;
					if (e.wheel.y < 0) zoom /= 1.1f;
					break;
				case SDL_EVENT_MOUSE_BUTTON_DOWN:
					if (e.button.button == SDL_BUTTON_LEFT) {
						dragging = true;
						lastMouseX = e.button.x;
						lastMouseY = e.button.y;
					} else if (e.button.button == SDL_BUTTON_RIGHT && pause) {
						step_simulation(num_particles, num_states, dt, max_velocity, target_energy, frameskip, potential_gain);
					} else if (e.button.button == SDL_BUTTON_RIGHT && !pause) {
						undither = !undither;
					}
					break;
				case SDL_EVENT_MOUSE_BUTTON_UP:
					if (e.button.button == SDL_BUTTON_LEFT) {
						dragging = false;
					}
					break;
				case SDL_EVENT_MOUSE_MOTION:
					if (dragging) {
						offsetX -= (e.motion.x - lastMouseX) / zoom;
						offsetY -= (e.motion.y - lastMouseY) / zoom;
						lastMouseX = e.motion.x;
						lastMouseY = e.motion.y;
					}
					break;
			}
		}
        // Run simulation step - d_particles updated in place in managed memory
		
		if (!pause) step_simulation(num_particles, num_states, dt, max_velocity, target_energy, frameskip, potential_gain);

        // Copy particle positions back to host vector for any CPU-side logic if needed
        memcpy(particles.data(), d_particles, sizeof(Particle) * num_particles);
		int num_visible = 0;
		std::vector<Particle> visible_particles;
		for (int i = 0; i < num_particles; ++i) {
			Particle& p = particles[i];
			int x = (int)((p.x - offsetX - width / 2.0f) * zoom + width / 2.0f);
			int y = (int)((p.y - offsetY - height / 2.0f) * zoom + height / 2.0f);
			if (x >= 0 && x < width && y >= 0 && y < height) {
				visible_particles.push_back(p);
				num_visible++;
			}
		}
		// Clear framebuffer to black
		memset(framebuffer.data(), 0, width * height * sizeof(uint32_t));
		memset(accum.data(), 0, width * height * sizeof(uint32_t));
		// Set each particle as a pixel
		if (undither) {
			dim3 threads(SMOOTH_BLOCK_SIZE, SMOOTH_BLOCK_SIZE);
			dim3 blocks((width + SMOOTH_BLOCK_SIZE - 1) / SMOOTH_BLOCK_SIZE, (height + SMOOTH_BLOCK_SIZE - 1) / SMOOTH_BLOCK_SIZE);
			std::copy(visible_particles.begin(), visible_particles.end(), d_visible);
			hipMemset(d_input, 0, width * height * sizeof(uint32_t));
			renderDensityBlend<<<blocks, threads>>>(d_input, d_visible, num_visible, width, height, num_states, offsetX, offsetY, zoom, particle_size, d_mass, d_target_energy);
			hipError_t err = hipGetLastError();
			if (err != hipSuccess) {
				std::cerr << "CUDA kernel error: " << hipGetErrorString(err) << std::endl;
			}
			hipDeviceSynchronize();
			hipMemcpy(accum.data(), d_input, width * height * sizeof(uint32_t), hipMemcpyDeviceToHost);
		} else {
			for (int i = 0; i < num_visible; ++i) {
				Particle& p = visible_particles[i];
				int x = (int)((p.x - offsetX - width / 2.0f) * zoom + width / 2.0f);
				int y = (int)((p.y - offsetY - height / 2.0f) * zoom + height / 2.0f);
				if (x >= 0 && x < width && y >= 0 && y < height) {
					float h = (float)p.state / (float)num_states;
					float s = fminf(1.0f, p.energy / mass[p.state] / target_energy / 2) * 3 / 4 + 0.25;
					float v = fminf(1.0f, fmaxf(0.0f, 1.0f - p.potential / (p.energy * p.energy))) * 3 / 4 + 0.25;
					float c = v * s;
					float x_col = c * (1 - fabsf(fmodf(h * 6.0f, 2) - 1));
					float m = v - c;

					float r_, g_, b_;

					int sector = (int)(h * 6);
					switch (sector) {
						case 0: r_ = c; g_ = x_col; b_ = 0; break;
						case 1: r_ = x_col; g_ = c; b_ = 0; break;
						case 2: r_ = 0; g_ = c; b_ = x_col; break;
						case 3: r_ = 0; g_ = x_col; b_ = c; break;
						case 4: r_ = x_col; g_ = 0; b_ = c; break;
						case 5: default: r_ = c; g_ = 0; b_ = x_col; break;
					}	
					for (int dx = -particle_size; dx <= particle_size; ++dx) {
						for (int dy = -particle_size; dy <= particle_size; ++dy) {
							float dist = 1.0f;
							if (particle_size > 0) {
								dist = 1.0f - sqrtf((dx * dx + dy * dy) / (2.0f * particle_size * particle_size) + 1e-6f);
							}
							float r = 0, g = 0, b = 0, a = 0;
							if ((x + dx) >= 0 && (x + dx) < width && (y + dy) >= 0 && (y + dy) < height) {
								unpackRGBA(accum[(y + dy) * width + x + dx], r, g, b, a);
								float a2_ = (a / 255.0f * (1.0f - dist) + dist);
								uint8_t a2 = (uint8_t)(fminf(255.0, fmaxf(0.0, a2_ * 255.0f)));
								uint8_t r2 = (uint8_t)(fminf(255.0, fmaxf(0.0, (r / 255.0f) * (a / 255.0f) * (1.0f - dist) + (r_ + m) * dist) * 255.0f / a2_));
								uint8_t g2 = (uint8_t)(fminf(255.0, fmaxf(0.0, (g / 255.0f) * (a / 255.0f) * (1.0f - dist) + (g_ + m) * dist) * 255.0f / a2_));
								uint8_t b2 = (uint8_t)(fminf(255.0, fmaxf(0.0, (b / 255.0f) * (a / 255.0f) * (1.0f - dist) + (b_ + m) * dist) * 255.0f / a2_));
								accum[(y + dy) * width + x + dx] = (a2 << 24) | (b2 << 16) | (g2 << 8) | r2;
							}
						}
					}
				}
			}
		}
		SDL_UpdateTexture(texture, nullptr, accum.data(), width * sizeof(uint32_t));
		
        SDL_RenderClear(renderer);
        SDL_RenderTexture(renderer, texture, nullptr, nullptr);
        SDL_RenderPresent(renderer);

        SDL_Delay(1000 / fps);
    }

    // Cleanup CUDA managed buffers
    free_cuda_managed_buffers();

    SDL_DestroyTexture(texture);
    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}